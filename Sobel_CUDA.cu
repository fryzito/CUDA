#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include "mypgm.h"

#define WIN_SIZE 3
#define N 512//16384



__global__ void kernel(unsigned char i[][N],unsigned char Img[][N],double mini,double maxi)
{
  printf("Primer Kernel%uc\n",i[0][0]);
  int dx=blockIdx.x*blockDim.x+threadIdx.x;
  int dy=blockIdx.y*blockDim.y+threadIdx.y;
  //printf("dx=(%d,%d,%d) dy=(%d,%d,%d)\n",blockIdx.x,blockDim.x,threadIdx.x,blockIdx.y,blockDim.y,threadIdx.y);
  double p = 0.0;  
  //  printf("%d %d\n",dx,dy);
  if(dx != 0 && dy != 0) {
    p += -1 * i[dx-1][dy-1];
    p += 0  * i[dx-1][dy];
    p += 1  * i[dx-1][dy+1];
    p += -2 * i[dx][dy-1];
    p += 0  * i[dx][dy];
    p += 2  * i[dx][dy+1];
    p += -1 * i[dx+1][dy-1];
    p += 0  * i[dx+1][dy];
    p += 1  * i[dx+1][dy+1];
    
    p = (double)MAX_BRIGHTNESS * (double)(p - mini) / (double)(maxi - mini);
    //  printf("%lf\n",p);
    //printf("%lf\n",p);
    Img[dy][dx] = (unsigned char)p;
  }
  
  
}



void move(unsigned char *a){

  for(int i=0;i<MAX_IMAGESIZE;i++){
    for(int j=0;j<MAX_IMAGESIZE;j++){
      image2[i][j]=0;
    }
  }
  
  for(int i=0;i<512;i++){
    for(int j=0;j<512;j++){
      image2[i][j]=a[i*N + j];
    }
  }
}

void getMaxMin(double &min,double &max){
  int weight[3][3] = {{ -1,  0,  1 },
                      { -2,  0,  2 },
                      { -1,  0,  1 }};
  double pixel_value;
  int x, y, i, j;  /* Loop variable */
  /* Maximum values calculation after filtering*/
  printf("Se esta procediendo a hallar la matriz\n\n");
  min = DBL_MAX;
  max = -DBL_MAX;
  for (y = 1; y < y_size1 - 1; y++) {
    for (x = 1; x < x_size1 - 1; x++) {
      pixel_value = 0.0;
      for (j = -1; j <= 1; j++) {
        for (i = -1; i <= 1; i++) {
          pixel_value += weight[j + 1][i + 1] * image1[y + j][x + i];
        }
      }
      if (pixel_value < min) min = pixel_value;
      if (pixel_value > max) max = pixel_value;
    }
  }
  if ((int)(max - min) == 0) {
    printf("No existe el archivo!!!\n\n");
    exit(1);
  }
}

int main(void){
  
  load_image_data( ); 

  ////////////////////////////////////////////////////
  // image1[][];
  // Reservar memoria en GPU
  unsigned char (*pA)[N],(*psobel)[N];
  int (*w)[3];
  hipMalloc((void**)&pA,(N*N)*sizeof(unsigned char));
  hipMalloc((void**)&psobel,(N*N)*sizeof(unsigned char));
  hipMalloc((void**)&w,(N*N)*sizeof(int));

  // Mover a device
  hipMemcpy(pA, image1, (N*N)*sizeof(unsigned char),hipMemcpyHostToDevice);


  const dim3 dimGrid(4,4);
  const dim3 dimBlock(16,16);
  double min=0,max=0;
  getMaxMin(min,max);
  printf("main %uc\n",image1[0][0]);
  kernel<<<dimGrid,dimBlock>>>(pA,psobel,min,max);
  printf("salio kernel\n");

  hipDeviceSynchronize();
  unsigned char* image3 =(unsigned char*)malloc(N*N*sizeof(unsigned char));
  hipMemcpy(image3, psobel, (N*N)*sizeof(unsigned char),hipMemcpyDeviceToHost);
  hipDeviceSynchronize();

  move(image3);
  // printf("0000000000000000000000000000000000000========================");
  //  for(int i=0;i<64;i++){
  //    for(int j=0;j<64;j++){
  //      printf("xx%d ",(int)image3[i*N + j]);
  //    }
  //  }
  
  //free resources
  hipFree(pA); 
  hipFree(psobel);
  
  //////////////////////////////////////////////////////
  
 
  x_size2 = x_size1;
  y_size2 = y_size1;
  save_image_data( ); 
  
  return 0;
}
